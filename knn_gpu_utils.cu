#include "hip/hip_runtime.h"
/**
 *
 * Implementations of CUDA kernels and functions used to compute the
 * k nearest neightbors.
 *
 * Problem hightlights: (Choosen when to program is called)
 * Maximum N = 2^20 data points with maximum dimension 128. 
 * Up to Q = 1000 queries with k = 1 to 8 neightbors for each query.
 *
 * Design choice: Every block has 128 threads, which is the maximum
 * data dimension. Thus, one block can compute the distance between
 * two vectors with equal or less than 128 dimensions. We will start
 * a grid of blocks with rows (y dimension) equal to the number of queries
 * (max 1000), an the number of columns (x dimension) will be analogus to
 * the number of data points. Because different GPUs tend to have different
 * global memory size, we will compute the distance between Q queries and
 * (at most) 2^16 data points in GPU (defined with the MAX_DATA_POINT 
 * global variable).
 * 
 * It is easy to change the code a little bit if there are more constraints,
 * or go to a previous version, if there is a larger global memory available
 * on the device. In general, it will be easy to fine-tune this code on any
 * GPU.
 *
 *
 * Previous versions avialable from github:
 * https://github.com/cNikolaou/cuKNN
 *
 *
 * Author: Christos Nikolaou
 * Date: August 2014
 *
 */


#include <stdio.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "knn_gpu_utils.h"
}

// NOTE: DIM_THREADS and MAX_THREADS should be choosen based on GPU's
// architecture.
// Number of threads per data point
#define DIM_THREADS 128
// Number of threads per block. Should be a multiple of DIM_THREADS, so that
// more than one data points will be computed from within a block.
#define MAX_THREADS 128
// Maximum number of blocks available in each dimension (GPU restriction)
#define MAX_BLOCKS 65535 

// Define the maximum number of points that will be used for the computation
// for each for-loop (in compute_distance_gpu() function). You can change 
// the value based on the available memory on your system. For each additional 
// data point you will need additional D*sizeof(double) memory space to sace
// the data point plus Q*D*sizeof(double) memory to save the computed distance.
#define MAX_DATA_POINTS 2

// Define the maximum number of queries that will be used in each loop of the
// for loop, when finding the minimum distances in selection_gpu() function.
// For each additional query there is a need for (approximately) 
// N*sizeof(double) additional memory space needed in the GPU.
#define MAX_QUERIES 1

// Define CUDA condition check.
#define CUDA_CHECK(condition) \
/* Code block avoids redefinition of hipError_t error */ \
do { \
  hipError_t error = condition; \
  if (error != hipSuccess) \
  printf("%s\n", hipGetErrorString(error)); \
} while (0)


// Print values of GPU arrays; used for debugging
void print_gpu_mat(const double *mat, const int length) {

  double *hostmat = (double*)malloc(length*sizeof(double));

  hipMemcpy(hostmat, mat, length*sizeof(double), hipMemcpyDeviceToHost);

  for (int i = 0; i < length; ++i) {
    printf("mat[%d] = %f\n", i, hostmat[i]);
  }

  free(hostmat);
}

// Print values of CPU arrays; used for debugging
void print_cpu_mat(const double *mat, const int length) {

  for (int i = 0; i < length; ++i) {
    printf("mat[%d] = %f\n", i, mat[i]);
  }

}

// Print device informations; used for debugging
void printDevProp(hipDeviceProp_t devProp) {
    
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ?"Yes" : "No"));
    return;

}

// Print information for all the available devices; used for debugging
void print_devices_data() {
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("CUDA Device Query...\n");
    printf("There are %d CUDA devices.\n", devCount);
 
    for (int i = 0; i < devCount; ++i)
    {
        // Get device properties
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printDevProp(devProp);
    }  
}

// Function to compute the difference between two vectors. 
__global__ void compute_dist(const double* data, const double* query, 
                             double* dist, const int D, const int N) {
  
  // Use shared memory for faster computation and reduction (in each block)
  __shared__ double Z[DIM_THREADS][MAX_THREADS/DIM_THREADS];
  
  // Thread index in block
  unsigned int tix = threadIdx.x;
  unsigned int tiy = threadIdx.y;
  
  // Index in current block for each 'data' matrix dimension
  unsigned int data_ind = threadIdx.x + threadIdx.y*D + 
                          blockIdx.x*blockDim.y*D;

  // Index in current grid line (y-dimesnion) for every query computation                          
  unsigned int query_ind = threadIdx.x + blockIdx.y*D;


  // Save the difference in the appropriate possition.
  double tmp = 0;

  // When thread's index is less than data's dimenension, then compute distance
  if (tix < D && data_ind < N*D) {
    tmp = data[data_ind] - query[query_ind];
  }

  Z[tix][tiy] = tmp * tmp;

  // Synchronize threads before reduction
  __syncthreads();

  // Perform reduction
  for (int offset = blockDim.x/2; offset > 0; offset >>=1) {

    if (tix < offset) {
      Z[tix][tiy] += Z[tix + offset][tiy];
    }
    __syncthreads();
  }

  // Save the sum at the appropriate position in the diff vector.
  if (tix == 0)// && data_ind < N*D) 
    dist[threadIdx.y + blockIdx.x*blockDim.y+ MAX_DATA_POINTS*blockIdx.y] 
          = Z[tix][tiy];

}


// Compute the euclidean between each D-dimensional row of 'data' matrix and 
// the D-dimensional row of 'queries' matrix.
extern "C"
void compute_distance_gpu(const double *data, const double *queries,
                          const int D, const int Q, const int N,
                          double *dist) {

  print_devices_data();

  // Define cuda error
  hipError_t cudaerr;

  // Define block size
  const int block_y_dim = MAX_THREADS/DIM_THREADS;
  dim3 blockSize(DIM_THREADS,block_y_dim);
 
  // Define grid size
  int num_blocks_x, num_blocks_y;

  // if number of 'data' D-dimensiona arrays are not equally divided by 
  // the number of data points per block (block_y_dim), then grid's
  // x dimension will have a block that doesn't compute the distance
  // between block_y_dim points.
/*  if (N%block_y_dim == 0)
    num_blocks_x = (N/MAX_DATA_POINTS)/block_y_dim;
  else
    num_blocks_x = (N/MAX_DATA_POINTS)/block_y_dim + 1;  
    
  num_blocks_y = Q; //= num_blocks_x/MAX_BLOCKS + 1;
*/
  // There will be Q rows and there will be computed 
  num_blocks_x = MAX_DATA_POINTS/block_y_dim;
  num_blocks_y = Q;
  dim3 gridSize(num_blocks_x,num_blocks_y);

  printf("Parameters when computing distance:\n");
  printf("Block size = (%d,%d), grid size = (%d,%d), D = %d, Q = %d, N = %d\n", 
          DIM_THREADS, block_y_dim, num_blocks_x, num_blocks_y, D, Q, N);
/**/

  // Define and allocate the device space that will hold the appropriate data
  double *deviceData, *deviceQueries, *deviceDist; 
  
//  printf("Maximum memory used during computations: %d\n", (D*MAX_DATA_POINTS+Q*D+Q*N)*sizeof(double));

//  printf("Allocating device memory for the data matrix.\n");
  CUDA_CHECK(hipMalloc((void**) &deviceData, 
                        MAX_DATA_POINTS*D*sizeof(double))); 
//  printf("Allocating device memory for the queries matrix.\n");
  CUDA_CHECK(hipMalloc((void**) &deviceQueries, Q*D*sizeof(double)));
//  printf("Allocating device memory for the distance matrix.\n");
  CUDA_CHECK(hipMalloc((void**) &deviceDist, Q*MAX_DATA_POINTS*sizeof(double)));

  double *tempDist = (double*) malloc(Q*MAX_DATA_POINTS*sizeof(double));
//  printf("Transfering 'data' matrix from host to device.\n");
//  CUDA_CHECK(hipMemcpy(deviceData, data, N*D*sizeof(double), 
//                        hipMemcpyHostToDevice));
//  printf("Transfering 'queries' matrix from host to device.\n");
  CUDA_CHECK(hipMemcpy(deviceQueries, queries, Q*D*sizeof(double), 
                        hipMemcpyHostToDevice));

/*  printf("--- All Data Matrix ---\n");
  print_cpu_mat(data, N*D);
  printf("--- Queries Matrix ---\n");
  print_gpu_mat(deviceQueries, Q*D);
*/
  int max_iterations = N/MAX_DATA_POINTS;

  if (N%MAX_DATA_POINTS != 0) {
    max_iterations++;
  }

//  printf("Max iterations %d\n", max_iterations);
  int offset;

  for (int iter = 0; iter < max_iterations; ++iter) {
    
//    printf("Transfering 'data' matrix from host to device. Iter = %d\n", iter);
    
    // Offset in dist matrix
    offset = iter * MAX_DATA_POINTS;
    // offset*D equals to the values
    CUDA_CHECK(hipMemcpy(deviceData, &data[offset*D], 
                          MAX_DATA_POINTS*D*sizeof(double),
                          hipMemcpyHostToDevice));
    
//    printf("--- Data Matrix ---\n");
//    print_gpu_mat(deviceData, MAX_DATA_POINTS*D);

//    printf("Call kernel for distance computation.\n");
    compute_dist<<<gridSize, blockSize>>>(deviceData,deviceQueries,
                                          deviceDist,D,N);
    // Check for kernel errors
    cudaerr = hipGetLastError();
    if (cudaerr != hipSuccess)
      printf("Error: %s\n", hipGetErrorString(cudaerr));


//    printf("Transfering 'dist' matrix from device to host.\n");
    CUDA_CHECK(hipMemcpy(tempDist, deviceDist, Q*MAX_DATA_POINTS*sizeof(double), 
                        hipMemcpyDeviceToHost));

/*    printf("--- Temp dist ---\n");
    print_gpu_mat(deviceDist, Q*MAX_DATA_POINTS);
    printf("--- Transfer temp dist ---\n");
*/
    for (int qi = 0; qi < Q; qi++) {
        for (int i = 0; i < MAX_DATA_POINTS; i++) {
  //        printf("tempDist[%d] = %f\n", 
  //                  i + qi*MAX_DATA_POINTS, tempDist[i + qi*MAX_DATA_POINTS]);
          dist[i + qi*N + iter*MAX_DATA_POINTS] = 
                                              tempDist[i + qi*MAX_DATA_POINTS];  
        } 
      } 
  }

/*  
  printf("--- Data Matrix ---\n");
  print_gpu_mat(deviceData, N*D);
  printf("--- Queries Matrix ---\n");
  print_gpu_mat(deviceQueries, Q*D);
*/  


  // Only for debugging purposes.
/*
  int max_qi = Q;
  // if N is greater than 10K, then print only the first 10K elements
  int max_i = (N > 10000) ? 10000 : N; 
  
  for (int qi = 0; qi < max_qi; qi++) {
    for (int i = 0; i < max_i; i++) {  
      printf("qi = %d, i = %d, dist = %f\n", qi, i, dist[qi*N + i]);
    }
  }
*/

  hipFree(deviceData); 
  hipFree(deviceQueries);
  hipFree(deviceDist);
}


/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */
/* ++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ */

__global__ void block_min(const double* dist, const int* index, const int N,
                          double* block_min_dist, int* block_min_indx) {
  
  unsigned int tix = threadIdx.x;
  unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
  unsigned int bix = blockIdx.x;

  // Use shared memory for faster computation and reduction (in each block)
  __shared__ double local_min_dist[MAX_THREADS];
  __shared__ double local_min_indx[MAX_THREADS];


  if (idx < N) {
    local_min_dist[tix] = dist[idx];
    local_min_indx[tix] = index[idx];
  } else {
    local_min_dist[tix] = DBL_MAX;
    local_min_indx[tix] = -1;
  }

  __syncthreads();


  for (int offset = blockDim.x/2; offset > 0; offset >>=1) {
    
    if (tix < offset) {

      if (local_min_dist[tix] > local_min_dist[tix + offset]) {
        local_min_dist[tix] = local_min_dist[tix + offset];
        local_min_indx[tix] = local_min_indx[tix + offset];
      }
    }
    __syncthreads();
  }

  // all blocks write their outcome
  if (tix == 0) {
    block_min_dist[bix] = local_min_dist[0];
    block_min_indx[bix] = local_min_indx[0];
  }
}

__global__ void total_min(const double* block_min_dist, 
                          const int* block_min_idx, 
                          const int n_blocks, 
                          int k, double* min_dist, int* min_idx) {

  unsigned int tix = threadIdx.x;
  
  __shared__ double local[MAX_THREADS];

  local[tix] = block_min_dist[tix];

  for (int offset = blockDim.x/2; offset > 0; offset >>=1) {
        
    if (tix < offset) {
      if (local[tix] > local[tix + offset])
        local[tix] = local[tix+offset];  
    }
    __syncthreads();
    
  }

  if (tix == 0) {
    min_dist[k] = block_min_dist[1];  
    min_idx[k] = block_min_dist[0];
  }
  
}

void compute_min() {
  
}

extern "C"
void selection_gpu(double* dist, double* NNdist, 
                   int* NNidx, int N, int Q, int k) {

  // ----- First phase reduction block and grid -----
  // GPU's block dimension
  int block_y_dim = 1;
  dim3 blockSize(MAX_THREADS, block_y_dim);

  // GPU's grid dimension; there will be MAX_QUERIES processed at the same time
  int num_blocks_x = N/MAX_THREADS + 1;
  int num_blocks_y = MAX_QUERIES;
  dim3 gridSize(num_blocks_x, num_blocks_y);

  // ----- Second phase reduction block and grid -----
  int block_y_dim_2 = 1;
  dim3 blockSize_2(MAX_THREADS, block_y_dim_2);

  // GPU's grid dimension; there will be MAX_QUERIES processed at the same time
  int num_blocks_x_2 = num_blocks_x/MAX_THREADS + 1;
  int num_blocks_y_2 = MAX_QUERIES;
  dim3 gridSize_2(num_blocks_x_2, num_blocks_y_2);

  // ----- Third phase reduction block and grid -----
  int block_y_dim_3 = 1;
  dim3 blockSize_3(MAX_THREADS, block_y_dim_3);

  // GPU's grid dimension; there will be MAX_QUERIES processed at the same time
  int num_blocks_x_3 = num_blocks_x_2/MAX_THREADS + 1;
  int num_blocks_y_3 = MAX_QUERIES;
  dim3 gridSize_3(num_blocks_x_3, num_blocks_y_3);


  printf("Parameters when selecting the k neighbors:\n");
  printf("Phase one: block size = (%d,%d), grid size = (%d,%d)\n", 
          MAX_THREADS, block_y_dim, num_blocks_x, num_blocks_y);
  printf("Phase two: block size = (%d,%d), grid size = (%d,%d)\n", 
          MAX_THREADS, block_y_dim_2, num_blocks_x_2, num_blocks_y_2);
  printf("Phase three: block size = (%d,%d), grid size = (%d,%d)\n", 
          MAX_THREADS, block_y_dim_3, num_blocks_x_3, num_blocks_y_3);
  
  if (num_blocks_x_3 > 1) {
    printf("ERROR; you need another reduction phase! Too many blocks!\n");  
  }

/**/

  // Array that holds the index. Used to find NNidx in the kernel.
  int idx[N];

  for (int i = 0; i < N; i++) {
    idx[i] = i;  
  }

  // CPU matrices that will be used to take 
  double *minDist = (double*) malloc(k*sizeof(double));
  int *minIdx = (int*) malloc(k*sizeof(int));

  // GPU arrays that will hold the data
  double *deviceDist, *deviceMinDist;
  int *deviceIdx, *deviceMinIdx;
  
  CUDA_CHECK(hipMalloc((void**) &deviceDist, MAX_QUERIES*N*sizeof(double)));
  CUDA_CHECK(hipMalloc((void**) &deviceMinDist, MAX_QUERIES*k*sizeof(double)));
  CUDA_CHECK(hipMalloc((void**) &deviceIdx, N*sizeof(int)));
  CUDA_CHECK(hipMalloc((void**) &deviceMinIdx, MAX_QUERIES*k*sizeof(int)));

  // Transfer indices in GPU
  CUDA_CHECK(hipMemcpy(deviceIdx, idx, N*sizeof(int), 
                        hipMemcpyHostToDevice));
/*
  // Print 
  printf("--- CPU distance matrix ---\n");
  print_cpu_mat(dist, Q*N);
*/

  double *deviceTmpMinDist, *deviceTmpMinDist_2;
  int *deviceTmpMinIdx, *deviceTmpMinIdx_2;
  
  CUDA_CHECK(hipMalloc((void**) &deviceTmpMinDist, 
                        MAX_QUERIES*num_blocks_x*sizeof(double)));
  CUDA_CHECK(hipMalloc((void**) &deviceTmpMinIdx, 
                        MAX_QUERIES*num_blocks_x*sizeof(int)));

  CUDA_CHECK(hipMalloc((void**) &deviceTmpMinDist_2, 
                        MAX_QUERIES*num_blocks_x_2*sizeof(double)));
  CUDA_CHECK(hipMalloc((void**) &deviceTmpMinIdx_2, 
                        MAX_QUERIES*num_blocks_x_2*sizeof(int)));


  int max_iterations = Q/MAX_QUERIES;

  for (int iter = 0; iter < max_iterations; ++iter) {
    
    for (int neighbor = 0; neighbor < k; ++neighbor) {
      CUDA_CHECK(hipMemcpy(deviceDist, &dist[iter*N*MAX_QUERIES], 
                            N*MAX_QUERIES*sizeof(double),
                            hipMemcpyHostToDevice));
/*    
      printf("--- GPU distance matrix ---\n");
      print_gpu_mat(deviceDist, N*MAX_QUERIES);
  */  
      block_min<<<gridSize, blockSize>>>(deviceDist, deviceIdx, N, 
                                           deviceTmpMinDist, deviceTmpMinIdx);

      block_min<<<gridSize_2, blockSize_2>>>(deviceTmpMinDist, 
                                             deviceTmpMinIdx, num_blocks_x, 
                                             deviceTmpMinDist_2, 
                                             deviceTmpMinIdx_2);

      block_min<<<gridSize_3, blockSize_3>>>(deviceTmpMinDist_2, 
                                             deviceTmpMinIdx_2, num_blocks_x_2, 
                                             deviceMinDist, deviceMinIdx);
/*
      printf("--- GPU minimum distance matrix ---\n");
      print_gpu_mat(deviceMinDist, MAX_QUERIES);
*/
      CUDA_CHECK(hipMemcpy(minDist, deviceMinDist, 
                            MAX_QUERIES*k*sizeof(double),
                            hipMemcpyDeviceToHost));    
        
      CUDA_CHECK(hipMemcpy(minIdx, deviceMinIdx, 
                            MAX_QUERIES*k*sizeof(int),
                            hipMemcpyDeviceToHost));
/*
      printf("--- GPU minimum index matrix ---\n");
      printf("Index = %d\n", minIdx[0]);
*/
      NNdist[iter*k + neighbor] = minDist[0];
      NNidx[iter*k + neighbor] = minIdx[0];
      dist[iter*N*MAX_QUERIES+minIdx[0]] = 500000;//DBL_MAX;
    }
  }

  hipFree(deviceDist);
  hipFree(deviceMinDist);
  hipFree(deviceIdx);
  hipFree(deviceMinIdx);
  hipFree(deviceTmpMinDist);
  hipFree(deviceTmpMinIdx);
  hipFree(deviceTmpMinDist_2);
  hipFree(deviceTmpMinIdx_2);

}

/**/
