#include "hip/hip_runtime.h"
/**
 *
 * Implementations of CUDA kernels and functions used to compute the
 * k nearest neightbors.
 *
 * Problem hightlits: Maximum N = 2^20 data points with maximum 
 * dimension 128. Up to Q = 1000 queries with k = 1:8 neightbors
 * for each query.
 *
 * Design choice: Every block has 128 threads, which is the maximum
 * data dimension. Thus, one block can compute the distance between
 * two vectors with equal or less than 128 dimensions. We will start
 * multiple blocks to compute the distance between one data point
 * and all queries. We will start queries->secondary_dim blocks.
 *
 * Author: Christos Nikolaou
 * Date: August 2014
 *
 */


#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "knn_gpu_utils.h"
}

// Number of threads per block.
#define DIM_THREADS 128
#define MAX_THREADS 512

// Define CUDA condition check.
#define CUDA_CHECK(condition) \
/* Code block avoids redefinition of hipError_t error */ \
do { \
hipError_t error = condition; \
if (error != hipSuccess) \
printf("%s\n", hipGetErrorString(error)); \
} while (0)


// Function to compute the difference between two vectors. 
__global__ void compute_diff(double* data, double* query, double* dist, int D, int N, int index) {
  
  // Use shared memory for faster computation and reduction
  __shared__ double Z[DIM_THREADS][MAX_THREADS/DIM_THREADS];
  unsigned int tix = threadIdx.x;
  unsigned int tiy = threadIdx.y;
/*  unsigned int i = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x +
                    threadIdx.x;
  */unsigned int ind = threadIdx.x + threadIdx.y*D + blockIdx.x*D*blockDim.y;
  
  // Save the difference in the appropriate possition.
  double tmp = 0;

  if (tix < D) {
    tmp = data[ind] - query[tix];
  }

  Z[tix][tiy] = tmp * tmp;

  __syncthreads();

  // Perform reduction
  for (int offset = blockDim.x/2; offset > 0; offset >>=1) {

    if (tix < offset) {
      Z[tix][tiy] += Z[tix + offset][tiy];
    }
    __syncthreads();
  }

  // Save the sum at the appropriate position in the diff vector.
  if (tix == 0) 
    dist[threadIdx.y + blockIdx.x*blockDim.y + N*index] = Z[0][tiy];

}

// Testing function
extern "C"
void say_hello() {
  printf("Hallo!!!\n");
  return;
}

// Print values of GPU arrays
void print_GPU_Mat(double *mat, int length) {

  double *hostmat = (double*)malloc(length*sizeof(double));

  hipMemcpy(hostmat, mat, length*sizeof(double), hipMemcpyDeviceToHost);

  for (int i = 0; i < length; ++i) {
    printf("mat[%d] = %f\n", i, hostmat[i]);
  }

  free(hostmat);
}

// Print values of CPU arrays 
void print_CPU_Mat(double *mat, int length) {

  for (int i = 0; i < length; ++i) {
    printf("mat[%d] = %f\n", i, mat[i]);
  }

}


// Compute the euclidean between the N-dimensional vectors X and Y.
extern "C"
void euclidean_distance(double *devData, double *devQueries, int D, int Q, 
                        int N, int index, double *devDist) {

  // Define cuda error
  hipError_t cudaerr;

  // Define block and grid size
  const int y_dim = MAX_THREADS/DIM_THREADS;
  dim3 blockSize(DIM_THREADS,y_dim);
  const int num_blocks = N/y_dim + 1;
  dim3 gridSize(num_blocks,1);
/*
  printf("Block size = (%d,%d), grid size = (%d,1), D = %d, Q = %, N = %d\n", 
          DIM_THREADS, y_dim, num_blocks, D, Q, N);
*/

/*
  printf("--- Print matrix devData ---\n");
  print_GPU_Mat(devData,N*D);
  printf("--- Print matrix devQueries ---\n");
  print_GPU_Mat(devQueries,D);
*/

  // Define the array to hold the computed difference between vectors

  compute_diff<<<gridSize, blockSize>>>(devData,devQueries,devDist,D,N,index);

  // Check for kernel errors
  cudaerr = hipGetLastError();
  if (cudaerr != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(cudaerr));

}



// Function that transfers the data to device memory
extern "C"
void compute_distance_gpu(double *data, double *queries, int D, int Q, int N,
                          double *dist) {

  // Define and allocate the device space that will hold the appropriate data
  double *deviceData, *deviceQueries, *deviceDist; 
  
  CUDA_CHECK(hipMalloc((void**) &deviceData, N*D*sizeof(double)));
  CUDA_CHECK(hipMalloc((void**) &deviceQueries, Q*D*sizeof(double)));
  CUDA_CHECK(hipMalloc((void**) &deviceDist, Q*N*sizeof(double)));

  CUDA_CHECK(hipMemcpy(deviceData, data, N*D*sizeof(double), 
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceQueries, queries, Q*D*sizeof(double), 
                        hipMemcpyHostToDevice));

  int i, j, qi;

  for (qi=0; qi<Q; qi++) {

    euclidean_distance(deviceData, &deviceQueries[qi*D], D, Q, N, qi, 
                        deviceDist);

  }

  CUDA_CHECK(hipMemcpy(dist, deviceDist, N*Q*sizeof(double), 
                        hipMemcpyDeviceToHost));
/*
  for(qi=0; qi<Q; qi++){
    for(i=0; i<N; i++){  
      printf("qi = %d, i = %d, dist = %f\n", qi, i, dist[qi*N + i]);
    }
  }
*/

  hipFree(deviceData); 
  hipFree(deviceQueries);
  hipFree(deviceDist);
}
